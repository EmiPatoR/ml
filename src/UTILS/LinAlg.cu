#include "LinAlg.hpp"

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
 void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for (int i = 0; i < nr_rows_A; ++i) {
        for (int j = 0; j < nr_cols_A; ++j) {
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

Eigen::MatrixXf ISAE_ML::UTILS::LinAlg::transpose(const Eigen::MatrixXf &A) {
    // Allocation and copy to GPU Memory
    thrust::device_vector<float> d_A(A.data(), A.data() + A.rows() * A.cols());
    thrust::device_vector<float> d_B((unsigned long) A.cols() * A.rows());
    thrust::device_vector<float> d_At((unsigned long) A.cols() * A.rows());

    // Allocation to CPU Memory
    thrust::device_vector<float> h_At((unsigned long) A.cols() * A.rows());

    // Pointers
    float* pd_A = thrust::raw_pointer_cast(&d_A[0]);
    float* pd_B = thrust::raw_pointer_cast(&d_B[0]);
    float* pd_At = thrust::raw_pointer_cast(&d_At[0]);
    float* ph_At = thrust::raw_pointer_cast(&h_At[0]);

    // Init handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alf = 1;
    const float bet = 0;
    const float* alpha = &alf;
    const float* beta = &bet;

    int m = (int) A.rows();
    int n = (int) A.cols();

    //TODO CORRECT NOT WORKING
    // Do the transposition
    hipblasSgeam(handle, HIPBLAS_OP_T,HIPBLAS_OP_T,n,m,alpha,pd_A,n,beta,pd_B,n,pd_At,n);

    std::cout << "TEST" << std::endl;
    // Copy from device to host
    thrust::copy(d_At.begin(), d_At.end(), h_At.begin());
    print_matrix(ph_At,A.rows(), A.cols());

    // Destroy Handle
    hipblasDestroy(handle);

    return Eigen::Map<Eigen::MatrixXf>(ph_At, A.cols(), A.rows());
}

Eigen::MatrixXf ISAE_ML::UTILS::LinAlg::matrixMult(const Eigen::MatrixXf& A, const Eigen::MatrixXf& B, bool transposeA,
                           bool transposeB){


    // Allocating and copying Matrices on GPU
    thrust::device_vector<float> d_A(A.data(), A.data() + A.rows() * A.cols());
    thrust::device_vector<float> d_B(B.data(), B.data() + B.rows() * B.cols());
    thrust::device_vector<float> d_C((unsigned long)A.rows() * B.cols());

    //Allocating result matrix on CPU
    thrust::host_vector<float> h_C ((unsigned long)A.rows() * B.cols());


    // Pointers to the Matrices
    float* pd_A = thrust::raw_pointer_cast(&d_A[0]);
    float* pd_B = thrust::raw_pointer_cast(&d_B[0]);
    float* pd_C = thrust::raw_pointer_cast(&d_C[0]);
    float* ph_C = thrust::raw_pointer_cast(&h_C[0]);

    // cuBLAS params
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    int m = (int) A.rows();
    int n = (int) B.cols();
    int k = (int) A.cols();

    int lda = m;
    int ldb = k;
    int ldc = m;

    // Init handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do we need to Transpose ?
    hipblasOperation_t trA = (transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N);
    hipblasOperation_t trB = (transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N);

    // Do the multiplication

    hipblasSgemm(handle, trA, trB, m, n, k, alpha, pd_A, lda,
                   pd_B, ldb, beta, pd_C, ldc);


    // Copy from device to host
    thrust::copy(d_C.begin(), d_C.end(), h_C.begin());


    // Destroy Handle
    hipblasDestroy(handle);


    // Return result
    return Eigen::Map<Eigen::MatrixXf>(ph_C, A.rows(), B.cols());
}


